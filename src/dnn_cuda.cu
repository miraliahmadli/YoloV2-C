
#include <hip/hip_runtime.h>
#include <iostream>
#include <algorithm>
#include <stdio.h>
using namespace std;

#define BLOCK_SIZE 16
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

static void HandleError(hipError_t err, const char *file, int line)
{
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString( err ), file, line);
        exit(EXIT_FAILURE);
    }
}

__global__ void add(int *a, int *b, int *c, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n)
        c[index] = a[index] + b[index];
    }

void print_mat(const char * name, int r, int c, double *m){
    printf("Printing %s\n", name);
    for (int i = 0; i < r; i++) {
        for (int j = 0; j < c; j++) {
            printf("%.2lf ", m[i * c + j]);
        }
        printf("\n");
    }
}

/*

    Leaky RELU

*/
__global__  void gpu_l_relu(double *res, int n, int k, int channel){
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int offset = blockIdx.z * n * k;
    if( col < k && row < n) 
    {
        int index = offset + row * k + col;
        if(res[index] < 0.0) res[index] *= 0.1;
    }
}

extern "C" {
    void leaky_relu(double *res, int n, int k, int channel){
        // Allocate memory space on the device 
        double *dev_res;
        hipMalloc((void **) &dev_res, sizeof(double)*n*k*channel);

        // copy matrix A and B from host to device memory
        hipMemcpy(dev_res, res, sizeof(double)*n*k*channel, hipMemcpyHostToDevice);

        unsigned int gridev_rows = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
        unsigned int gridev_cols = (k + BLOCK_SIZE - 1) / BLOCK_SIZE;
        unsigned int channels = channel;
        dim3 dimGrid(gridev_cols, gridev_rows, channels);
        dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    
        // Launch kernel 
        gpu_l_relu<<<dimGrid, dimBlock>>>(dev_res, n, k, channel);    

        // Transefr results from device to host 
        hipMemcpy(res, dev_res, sizeof(double)*n*k*channel, hipMemcpyDeviceToHost);
        hipDeviceSynchronize();

        // free memory
        hipFree(dev_res);
    }
}


/*

    Batch Norm

*/
__global__ void b_norm (double *res, double *mean, double *gamma, 
                    double *variance, double epsilon, int n, int oc){
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int size = n*oc;
    if (index < size){
        int col = index % oc;
        double divisor = sqrt(variance[col] + epsilon);
        double divident = (res[index] - mean[col]) * gamma[col];
        res[index] = divident / divisor;
    }
}

extern "C" {
    void batch_norm(double *res, double *mean, double *gamma, 
                    double *variance, double epsilon, int n, int oc){
        double *dev_res, *dev_mean, *dev_gamma, *dev_variance;
        int size1 = oc * sizeof(double);
        int size2 = n * size1;

        // allocate the memory on the GPU
        HANDLE_ERROR( hipMalloc( (void**)&dev_res, size2 ) );

        HANDLE_ERROR( hipMalloc( (void**)&dev_mean, size1) );
        HANDLE_ERROR( hipMalloc( (void**)&dev_gamma, size1 ) );
        HANDLE_ERROR( hipMalloc( (void**)&dev_variance, size1 ) );
        
        // HANDLE_ERROR( cudaMalloc( (void **)&dev_epsilon, sizeof(double) ) );
        
        // copy the arrays to the GPU
        HANDLE_ERROR( hipMemcpy( dev_res, res, size2, hipMemcpyHostToDevice ) );

        HANDLE_ERROR( hipMemcpy( dev_mean, mean, size1, hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy( dev_gamma, gamma, size1, hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy( dev_variance, variance, size1, hipMemcpyHostToDevice ) );

        // Kernel invocation
        b_norm<<<(size2 + size1-1) / size1, size1>>>(dev_res, dev_mean, 
            dev_gamma, dev_variance, epsilon, n, oc);

        // copy the arrays back from the GPU to the CPU
        HANDLE_ERROR( hipMemcpy( res, dev_res, size2, hipMemcpyDeviceToHost ) );

        // free the memory allocated on the GPU
        hipFree( dev_res );
        hipFree( dev_mean );
        hipFree( dev_gamma );
        hipFree( dev_variance );
    }
}

/*

    Add Bias

*/

__global__ void gpu_add_bias (double *res, double *bias, int n, int k, int channel){
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int offset = blockIdx.z * n * k;
    if( col < k && row < n) 
    {
        res[offset + row * k + col] += bias[blockIdx.z];
    }
}

extern "C" {
    void add_bias(double * C, double * bias, int n, int k, int channel){
        // Allocate memory space on the device 
        double *dev_b, *dev_c;
        hipMalloc((void **) &dev_b, sizeof(double)*channel);
        hipMalloc((void **) &dev_c, sizeof(double)*n*k*channel);

        // copy matrix A and B from host to device memory
        hipMemcpy(dev_b, bias, sizeof(double)*channel, hipMemcpyHostToDevice);
        hipMemcpy(dev_c, C, sizeof(double)*n*k*channel, hipMemcpyHostToDevice);

        unsigned int gridev_rows = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
        unsigned int gridev_cols = (k + BLOCK_SIZE - 1) / BLOCK_SIZE;
        unsigned int channels = channel;
        dim3 dimGrid(gridev_cols, gridev_rows, channels);
        dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    
        // Launch kernel 
        gpu_add_bias<<<dimGrid, dimBlock>>>(dev_c, dev_b, n, k, channel);    

        // Transefr results from device to host 
        hipMemcpy(C, dev_c, sizeof(double)*n*k*channel, hipMemcpyDeviceToHost);
        hipDeviceSynchronize();

        // free memory
        hipFree(dev_b);
        hipFree(dev_c);
    }
}

/*

    MAX Pool

*/

/*

    Convolution

*/
__global__ void gpu_multABtoC(double *a,double *b, double *c, int m, int n, int k)
{ 
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    double sum = 0.0;
    if( col < k && row < m) 
    {
        for(int i = 0; i < n; i++) 
        {
            sum += a[row * n + i] * b[i * k + col];
        }
        c[row * k + col] = sum;
    }
} 

extern "C"{
    void conv2d(double *C, double *A, double *B, int m, int n, int k)
    {
        // Allocate memory space on the device 
        double *dev_a, *dev_b, *dev_c;
        hipMalloc((void **) &dev_a, sizeof(double)*m*n);
        hipMalloc((void **) &dev_b, sizeof(double)*n*k);
        hipMalloc((void **) &dev_c, sizeof(double)*m*k);

        // copy matrix A and B from host to device memory
        hipMemcpy(dev_a, A, sizeof(double)*m*n, hipMemcpyHostToDevice);
        hipMemcpy(dev_b, B, sizeof(double)*n*k, hipMemcpyHostToDevice);
        hipMemcpy(dev_c, C, sizeof(double)*m*k, hipMemcpyHostToDevice);

        unsigned int gridev_rows = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
        unsigned int gridev_cols = (k + BLOCK_SIZE - 1) / BLOCK_SIZE;
        dim3 dimGrid(gridev_cols, gridev_rows);
        dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    
        // Launch kernel 
        gpu_multABtoC<<<dimGrid, dimBlock>>>(dev_a, dev_b, dev_c, m, n, k);    

        // Transefr results from device to host 
        hipMemcpy(C, dev_c, sizeof(double)*m*k, hipMemcpyDeviceToHost);
        hipDeviceSynchronize();

        // free memory
        hipFree(dev_a);
        hipFree(dev_b);
        hipFree(dev_c);
    }
}